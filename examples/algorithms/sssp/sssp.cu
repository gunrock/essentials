#include <gunrock/algorithms/sssp.hxx>
#include "sssp_cpu.hxx"  // Reference implementation

using namespace gunrock;
using namespace memory;

void test_sssp(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  csr_t csr;
  std::string filename = argument_array[1];

  if (util::is_market(filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    csr.from_coo(mm.load(filename));
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );  // supports row_indices and column_offsets (default = nullptr)

  // --
  // Params and memory allocation
  srand(time(NULL));

  vertex_t n_vertices = G.get_number_of_vertices();
  vertex_t single_source = 483998; //1549256;  // rand() % n_vertices;
  std::cout << "Single Source = " << single_source << std::endl;

  // --
  // GPU Run

  /// An example of how one can use std::shared_ptr to allocate memory on the
  /// GPU, using a custom deleter that automatically handles deletion of the
  /// memory.
  // std::shared_ptr<weight_t> distances(
  //     allocate<weight_t>(n_vertices * sizeof(weight_t)),
  //     deleter_t<weight_t>());
  // std::shared_ptr<vertex_t> predecessors(
  //     allocate<vertex_t>(n_vertices * sizeof(vertex_t)),
  //     deleter_t<vertex_t>());

  thrust::device_vector<weight_t> distances(n_vertices);
  thrust::device_vector<vertex_t> predecessors(n_vertices);

  float gpu_elapsed = 0.0f;
  int num_runs = 5;

  for (auto i = 0; i < num_runs; i++)
    gpu_elapsed += gunrock::sssp::run(G, single_source, distances.data().get(),
                                      predecessors.data().get());

  gpu_elapsed /= num_runs;

  // --
  // CPU Run

  thrust::host_vector<weight_t> h_distances(n_vertices);
  thrust::host_vector<vertex_t> h_predecessors(n_vertices);

  float cpu_elapsed = sssp_cpu::run<csr_t, vertex_t, edge_t, weight_t>(
      csr, single_source, h_distances.data(), h_predecessors.data());

  int n_errors =
      util::compare(distances.data().get(), h_distances.data(), n_vertices);

  // --
  // Log + Validate

  print::head(distances, 40, "GPU distances");
  print::head(h_distances, 40, "CPU Distances");

  std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
  std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  std::cout << "Number of errors : " << n_errors << std::endl;
}

int main(int argc, char** argv) {
  test_sssp(argc, argv);
}
